
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

__global__ void matrix_multiply_with_race(float *A, float *B, float *C, int num_A_rows, int num_A_cols, int num_B_cols) {
    int block_id = blockIdx.y * gridDim.x + blockIdx.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);

    //int num_threads_for_C = blockDim.x * blockDim.y / 16;   // Sixteen threads for one element of C
    int elem_id = thread_id / 16;                           // Element ID of C this thread group works on
    int part = thread_id % 16;                              // Part of the computation assigned to this thread

    if (elem_id < num_A_rows * num_B_cols) {
        int element_row = elem_id / num_B_cols;
        int element_col = elem_id % num_B_cols;
        float sum = 0.0f;

        // Calculate start and end indices based on 'part'
        int start = part * (num_A_cols / 16);
        int end = (part == 15) ? num_A_cols : (part + 1) * (num_A_cols / 16);

        for (int i = start; i < end; i++) {
            sum += A[element_row * num_A_cols + i] * B[i * num_B_cols + element_col];
        }

        atomicAdd(&C[element_row * num_B_cols + element_col], sum);
    }
}

void initializeMatrix(float* matrix, int numRows, int numCols) {
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            matrix[i * numCols + j] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        }
    }
}

void runMatrixMultiplication(int num_A_rows, int num_A_cols, int num_B_cols) {
    dim3 threads_per_block(32, 32);
    dim3 num_blocks((num_B_cols + threads_per_block.x - 1) / threads_per_block.x,
                    (num_A_rows + threads_per_block.y - 1) / threads_per_block.y);

    std::cout << "Matrix size: " << num_A_rows << "x" << num_B_cols << std::endl;
    std::cout << "Number of blocks: " << num_blocks.x << " " << num_blocks.y << std::endl;
    std::cout << "Number of threads per block: " << threads_per_block.x << " " << threads_per_block.y << std::endl;

    float *A, *B, *C;
    float *A_gpu, *B_gpu, *C_gpu;
    A = (float *) malloc(num_A_rows * num_A_cols * sizeof(float));
    B = (float *) malloc(num_A_cols * num_B_cols * sizeof(float));
    C = (float *) malloc(num_A_rows * num_B_cols * sizeof(float));

    std::srand(42);

    // Initialize matrices A and B with random values
    initializeMatrix(A, num_A_rows, num_A_cols);
    initializeMatrix(B, num_A_cols, num_B_cols);

    hipMalloc((void **) &A_gpu, num_A_rows * num_A_cols * sizeof(float));
    hipMalloc((void **) &B_gpu, num_A_cols * num_B_cols * sizeof(float));
    hipMalloc((void **) &C_gpu, num_A_rows * num_B_cols * sizeof(float));

    hipMemcpy(A_gpu, A, num_A_rows * num_A_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, num_A_cols * num_B_cols * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    matrix_multiply_with_race<<<num_blocks, threads_per_block>>>(A_gpu, B_gpu, C_gpu, num_A_rows, num_A_cols, num_B_cols);

    hipDeviceSynchronize();

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, C_gpu, num_A_rows * num_B_cols * sizeof(float), hipMemcpyDeviceToHost);

    // Print the resulting matrix C
    std::cout << "Matrix C (Result):" << std::endl;
    for (int i = 0; i < 10; i++) { // Print only the first 10 rows for brevity
        for (int j = 0; j < 10; j++) { // Print only the first 10 columns for brevity
            std::cout << C[i * num_B_cols + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Time taken for matrix multiplication: " << milliseconds / 1000 << " seconds" << std::endl;

    // Free device memory
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);

    // Free host memory
    free(A);
    free(B);
    free(C);
}

int main() {
    runMatrixMultiplication(1000, 1000, 1000);
    runMatrixMultiplication(2500, 2500, 2500);
    runMatrixMultiplication(5000, 5000, 5000);

    return 0;
}
