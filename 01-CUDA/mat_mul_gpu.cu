
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

// Kernel function to perform matrix multiplication on the GPU
__global__ void matrix_multiply(float *A, float *B, float *C, int num_A_rows, int num_A_cols, int num_B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_A_rows && col < num_B_cols) {
        float sum = 0.0f;
        for (int i = 0; i < num_A_cols; i++) {
            sum += A[row * num_A_cols + i] * B[i * num_B_cols + col];
        }
        C[row * num_B_cols + col] = sum;
    }
}

// Function to initialize a matrix with random values
void initializeMatrix(float* matrix, int numRows, int numCols) {
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            matrix[i * numCols + j] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        }
    }
}

void runMatrixMultiplication(int num_A_rows, int num_A_cols, int num_B_cols) {
   
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((num_B_cols + threads_per_block.x - 1) / threads_per_block.x,
                    (num_A_rows + threads_per_block.y - 1) / threads_per_block.y);

    std::cout << "Number of blocks: " << num_blocks.x << " " << num_blocks.y << std::endl;
    std::cout << "Number of threads per block: " << threads_per_block.x << " " << threads_per_block.y << std::endl;

    float *A, *B, *C;
    float *A_gpu, *B_gpu, *C_gpu;
    A = (float *)malloc(num_A_rows * num_A_cols * sizeof(float));
    B = (float *)malloc(num_A_cols * num_B_cols * sizeof(float));
    C = (float *)malloc(num_A_rows * num_B_cols * sizeof(float));

    std::srand(42);

    // Initialize matrices A and B with random values
    initializeMatrix(A, num_A_rows, num_A_cols);
    initializeMatrix(B, num_A_cols, num_B_cols);

    hipMalloc((void **)&A_gpu, num_A_rows * num_A_cols * sizeof(float));
    hipMalloc((void **)&B_gpu, num_A_cols * num_B_cols * sizeof(float));
    hipMalloc((void **)&C_gpu, num_A_rows * num_B_cols * sizeof(float));

    hipMemcpy(A_gpu, A, num_A_rows * num_A_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, num_A_cols * num_B_cols * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch the matrix multiplication kernel
    matrix_multiply<<<num_blocks, threads_per_block>>>(A_gpu, B_gpu, C_gpu, num_A_rows, num_A_cols, num_B_cols);
    hipDeviceSynchronize();

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, C_gpu, num_A_rows * num_B_cols * sizeof(float), hipMemcpyDeviceToHost);

    // Print the resulting matrix C
    std::cout << "Matrix C (Result):" << std::endl;
    for (int i = 0; i < 10; i++) { // Print only the first 10 rows for brevity
        for (int j = 0; j < 10; j++) { // Print only the first 10 columns for brevity
            std::cout << C[i * num_B_cols + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Time taken for matrix multiplication: " << milliseconds / 1000 << " seconds" << std::endl;

    // Free device memory
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);

    // Free host memory
    free(A);
    free(B);
    free(C);
}

int main() {
    runMatrixMultiplication(1000, 1000, 1000);
    runMatrixMultiplication(2500, 2500, 2500);
    runMatrixMultiplication(5000, 5000, 5000);
    return 0;
}
